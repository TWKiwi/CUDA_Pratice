#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <ostream>
#include <iostream>
#include <thread>
#include <string>
#define DATA_SIZE 1048576
#define BLOCK_NUM 32
#define THREAD_NUM 256
#define MODE 0

std::string CMD = "";
int data[DATA_SIZE];
bool InitCUDA();
void GenerateNumbers(int *number, int size);
void matgen(float* a, int lda, int n);
void matmult(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n);
void ArrayCompute();
void ArrayCompute_multiple_threads();
void ArrayCompute_multiple_threads_continuous_access();
void ArrayCompute_multiple_threads_blocks_continuous_access();
void ArrayCompute_shared_multiple_threads_blocks_continuous_access();
void ArrayCompute_shared_multiple_threads_blocks_continuous_access_treesum();
void ArrayCompute_shared_multiple_threads_blocks_continuous_access_better_treesum();

__global__ static void sumOfSquares(int *num, int* result);
__global__ static void sumOfSquares_multiple_threads(int *num, int* result);
__global__ static void sumOfSquares_multiple_threads_continuous_access(int *num, int* result);
__global__ static void sumOfSquares_multiple_threads_blocks_continuous_access(int *num, int* result);
__global__ static void sumOfSquares_shared_multiple_threads_blocks_continuous_access(int *num, int* result);
__global__ static void sumOfSquares_shared_multiple_threads_blocks_continuous_access_treesum(int *num, int* result);
__global__ static void sumOfSquares_shared_multiple_threads_blocks_continuous_access_better_treesum(int *num, int* result);

void FloatArrayMultiCompute(int n);
clock_t matmultCUDA(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n);
__global__ static void matMultCUDA(const float* a, size_t lda, const float* b, size_t ldb, float* c, size_t ldc, int n);

void FloatArrayMultiCompute_KSF(int n);
clock_t matmultCUDA_KSF(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n);
__global__ static void matMultCUDA_KSF(const float* a, size_t lda, const float* b, size_t ldb, float* c, size_t ldc, int n);

void FloatArrayMultiCompute_KSF_shared_pitch(int n);
clock_t matmultCUDA_KSF_shared_pitch(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n);
__global__ static void matMultCUDA_KSF_shared_pitch(const float* a, size_t lda, const float* b, size_t ldb, float* c, size_t ldc, int n);

void inverse_matrix(int n);




///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////                                        //////////////////////////////////////////
/////////////////////////////////////                  ���C���K              //////////////////////////////////////////
/////////////////////////////////////                                        //////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
hipDeviceProp_t prop;

void func_cmd()
{
	std::string cmd = CMD;
	system(cmd.c_str());
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////                                        //////////////////////////////////////////
/////////////////////////////////////                  �D�i�J�I              //////////////////////////////////////////
/////////////////////////////////////                                        //////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


int main()
{
	if (MODE == 0)
	{

		if (InitCUDA())
		{
			ArrayCompute();
			printf("(1).%dThreads��}����\n", THREAD_NUM);
			ArrayCompute_multiple_threads();
			printf("(2)(3).%dThreads �s��O����s������\n", THREAD_NUM);
			ArrayCompute_multiple_threads_continuous_access();
			printf("(4).%dThreads %dBlocks �s��O����s������\n", THREAD_NUM, BLOCK_NUM);
			ArrayCompute_multiple_threads_blocks_continuous_access();
			printf("(5).Shared Memory %dThreads %dBlocks �s��O����s������\n", THREAD_NUM, BLOCK_NUM);
			ArrayCompute_shared_multiple_threads_blocks_continuous_access();
			printf("(6).TreeSum alg. Shared Memory %dThreads %dBlocks �s��O����s������\n", THREAD_NUM, BLOCK_NUM);
			ArrayCompute_shared_multiple_threads_blocks_continuous_access_treesum();
			printf("(7).��}TreeSum alg. Shared Memory %dThreads %dBlocks �s��O����s������\n", THREAD_NUM, BLOCK_NUM);
			ArrayCompute_shared_multiple_threads_blocks_continuous_access_better_treesum();
			printf("//////////////FLOAT////////////////\n");
			printf("/////////�B�I�ƪ��x�}���k//////////\n");
			int n = 0;
			printf("��J�x�}�j�p(�@�ӥ����):");
			scanf_s("%d", &n);
			printf("(1).�@�목��\n");
			FloatArrayMultiCompute(n);
			printf("(2).Kahan'sSummation Formula��}\n");
			FloatArrayMultiCompute_KSF(n);
			printf("(3).KSF��} Shared Memory Pitch\n");
			FloatArrayMultiCompute_KSF_shared_pitch(n);
			printf("(4).�ϯx�}\n");
			inverse_matrix(3);
			
		}

		printf("\nDone!");
		system("pause");
		return 0;
	}
	else
	{
		/*
		 * ���ư�����h�Ұ�exe�ĪG�񪽱���system()�n�A���p�O�]�������ΥD������s��Ұ����ε{�������Y�A�t���q�L���F�C
		 * �o�O�ڪ��q���C
		 */
		CMD = "cd ..\\Multi_Window_Display && start CUDA.exe && start CUDA_m_t.exe && start CUDA_m_t_c_a_(256).exe && start CUDA_m_t_c_a_(512).exe && start CUDA_m_t_b_c_a.exe && start CUDA_s_m_t_b_c_a.exe && start CUDA_s_m_t_b_c_a_t.exe && start CUDA_s_m_t_b_c_a_b_t.exe";
		std::thread thread(func_cmd);
		thread.join();
		
		return 0;
	}
	
	

	
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////                                        //////////////////////////////////////////
/////////////////////////////////////                  �򥻸�T              //////////////////////////////////////////
/////////////////////////////////////                                        //////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


bool InitCUDA()
{
	int count;

	hipGetDeviceCount(&count);
	if (count == 0)
	{
		fprintf(stderr, "No Device!!");
		return false;
	}

	int i;
	for (i = 0; i < count; i++)
	{
		
		if (hipGetDeviceProperties(&prop, i) == hipSuccess)
		{
			if (prop.major >= 1)
			{
				printf("��%d�i��d�W�� -> %s\n", count, prop.name);
				printf("--CUDA���� -> %d.%d\n", prop.major, prop.minor);
				char msg[256];
				sprintf_s(msg, "--�`Global Memory -> %.0f MBytes (%llu bytes)\n",
					(float)prop.totalGlobalMem / 1048576.0f, (unsigned long long) prop.totalGlobalMem);
				printf("%s", msg);
				printf("--%2d �� Multiprocessors\n", prop.multiProcessorCount);
				printf("--�C��Multiprocessors�̳̤j������ƶq:%d\n", prop.maxThreadsPerMultiProcessor);
				printf("--�C��Block�̳̤j������ƶq:%d\n", prop.maxThreadsPerBlock);
				printf("--GPU �̤j�ɯ�: %.0f MHz (%0.2f GHz)\n", prop.clockRate * 1e-3f, prop.clockRate * 1e-6f);
				printf("--�̤j�O����ɯ�: %.0f Mhz (%0.2f GHz)\n", prop.memoryClockRate * 1e-3f, prop.memoryClockRate * 1e-6f);
				break;
			}
		}
	}

	if (i == count)
	{
		fprintf(stderr, "No device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////                                        //////////////////////////////////////////
/////////////////////////////////////                  �����                //////////////////////////////////////////
/////////////////////////////////////                  ����M                //////////////////////////////////////////
/////////////////////////////////////                                        //////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

/*
 * �ͶüƦ�C
 */
void GenerateNumbers(int *number, int size)
{
	for (int i = 0; i < size; i++)
	{
		number[i] = rand() % 10;
	}
}

/*
 * 1 Thread����
 */
void ArrayCompute()
{
	float timeValue;


	//-----------------------------------------------
	
	GenerateNumbers(data, DATA_SIZE);
	int* gpudata, *result;
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int));
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);

	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);

	sumOfSquares << <1, 1, 0 >> >(gpudata, result);

	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);

	int sum;
	hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	
	printf("--sum (GPU): %d\n", sum);
	printf("--����ɶ� (GPU): %f\n", float(timeValue) / CLOCKS_PER_SEC);
	//-----------------------------------------------

	sum = 0;
	clock_t cpu_time = clock();
	for (int i = 0; i < DATA_SIZE; i++) {
		sum += data[i] * data[i];
	}
	printf("--sum (CPU): %d\n", sum);
	printf("--����ɶ� (CPU): %f\n", float(clock() - cpu_time) / CLOCKS_PER_SEC);

}
/*
* �쪩����[�`�{��
*/
__global__ static void sumOfSquares(int *num, int* result)
{
	int sum = 0;
	int i;
	for (i = 0; i < DATA_SIZE; i++) {
		sum += num[i] * num[i];
	}
	*result = sum;
}
/*
 * Multiple Threads����
 */
void ArrayCompute_multiple_threads()
{
	float timeValue;
	//-----------------------------------------------
	int* gpudata, *result;
	
	GenerateNumbers(data, DATA_SIZE);
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int)* THREAD_NUM);
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);

	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);

	sumOfSquares_multiple_threads << <1, THREAD_NUM, 0 >> >(gpudata, result);

	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);

	int sum[THREAD_NUM];
	hipMemcpy(&sum, result, sizeof(int)* THREAD_NUM, hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);

	//-----------------------------------------------
	int final_sum = 0;
	for (int i = 0; i < THREAD_NUM; i++) {
		final_sum += sum[i];
	}
	float clock_cycle = prop.clockRate * 1e-3f * (float(timeValue) / CLOCKS_PER_SEC); 
	float memory_bandwidth = 4 / (float(timeValue) / CLOCKS_PER_SEC); // �u�A�Ω�32�줸��ƫe�� (1024 * 1024 * 32(bit)) / 8(bit -> byte) * 1024(byte -> kb) * 1024(kb -> mb)
	printf("--sum (GPU): %d\n", final_sum);
	printf("--����ɶ� (GPU): %f | �ɯ�: %fMHz | �O�����W�e:%f MB/s\n", float(timeValue) / CLOCKS_PER_SEC, clock_cycle, memory_bandwidth);
}
/*
* ��}�ᥭ��[�`�{��
* multiple threads blocks
*/
__global__ static void sumOfSquares_multiple_threads(int *num, int* result)
{
	const int tid = threadIdx.x;
	const int size = DATA_SIZE / THREAD_NUM;
	int sum = 0;
	int i;
	for (i = tid * size; i < (tid + 1) * size; i++) {
		sum += num[i] * num[i];
	}
	result[tid] = sum;
}
/*
* Multiple Threads����
* �s��O����s������
*/
void ArrayCompute_multiple_threads_continuous_access()
{
	float timeValue;
	//-----------------------------------------------
	int* gpudata, *result;
	
	GenerateNumbers(data, DATA_SIZE);
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int)* THREAD_NUM);
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);

	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);

	sumOfSquares_multiple_threads_continuous_access << <1, THREAD_NUM, 0 >> >(gpudata, result);

	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);

	int sum[THREAD_NUM];
	hipMemcpy(&sum, result, sizeof(int)* THREAD_NUM, hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);

	//-----------------------------------------------
	int final_sum = 0;
	for (int i = 0; i < THREAD_NUM; i++) {
		final_sum += sum[i];
	}
	float clock_cycle = prop.clockRate * 1e-3f * (float(timeValue) / CLOCKS_PER_SEC);
	float memory_bandwidth = 4 / (float(timeValue) / CLOCKS_PER_SEC); // �u�A�Ω�32�줸��ƫe�� (1024 * 1024 * 32(bit)) / 8(bit -> byte) * 1024(byte -> kb) * 1024(kb -> mb)
	printf("--sum (GPU): %d\n", final_sum);
	printf("--����ɶ� (GPU): %f | �ɯ�: %fMHz | �O�����W�e:%f MB/s\n", float(timeValue) / CLOCKS_PER_SEC, clock_cycle, memory_bandwidth);
}
/*
* ��}�ᥭ��[�`�{��
* multiple threads �s��O����s��
*/
__global__ static void sumOfSquares_multiple_threads_continuous_access(int *num, int* result)
{
	const int tid = threadIdx.x;
	int sum = 0;
	int i;
	for (i = tid; i < DATA_SIZE; i += THREAD_NUM) {
		sum += num[i] * num[i];
	}
	result[tid] = sum;
}
/*
* Multiple Threads Blocks����
* �s��O����s������
*/
void ArrayCompute_multiple_threads_blocks_continuous_access()
{
	
	float timeValue;
	//-----------------------------------------------
	int* gpudata, *result;
	
	GenerateNumbers(data, DATA_SIZE);
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int)* THREAD_NUM * BLOCK_NUM);
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);

	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);

	sumOfSquares_multiple_threads_blocks_continuous_access << <BLOCK_NUM, THREAD_NUM, 0 >> >(gpudata, result);
	
	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);

	int sum[THREAD_NUM * BLOCK_NUM];
	hipMemcpy(&sum, result, sizeof(int)* THREAD_NUM * BLOCK_NUM, hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);
	//-----------------------------------------------
	int final_sum = 0;
	for (int i = 0; i < THREAD_NUM * BLOCK_NUM; i++) {
		final_sum += sum[i];
	}
	
	float clock_cycle = prop.clockRate * 1e-3f * (float(timeValue) / CLOCKS_PER_SEC);
	float memory_bandwidth = 4 / (float(timeValue) / CLOCKS_PER_SEC); // �u�A�Ω�32�줸��ƫe�� (1024 * 1024 * 32(bit)) / 8(bit -> byte) * 1024(byte -> kb) * 1024(kb -> mb)
	printf("--sum (GPU): %d\n", final_sum);
	printf("--����ɶ� (GPU): %f | �ɯ�: %fMHz | �O�����W�e:%f MB/s\n", float(timeValue) / CLOCKS_PER_SEC, clock_cycle, memory_bandwidth);
}
/*
* ��}�ᥭ��[�`�{��
* multiple threads blocks
*/
__global__ static void sumOfSquares_multiple_threads_blocks_continuous_access(int *num, int* result)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int sum = 0;
	int i;
	for (i = bid * THREAD_NUM + tid; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM) {
		sum += num[i] * num[i];
	}
	result[bid * THREAD_NUM + tid] = sum;
}
/*
* Shared Multiple Threads Blocks����
* �s��O����s������
*/
void ArrayCompute_shared_multiple_threads_blocks_continuous_access()
{

	float timeValue;
	//-----------------------------------------------
	int* gpudata, *result;
	
	GenerateNumbers(data, DATA_SIZE);
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int)* THREAD_NUM * BLOCK_NUM);
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);

	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);

	sumOfSquares_shared_multiple_threads_blocks_continuous_access << <BLOCK_NUM, THREAD_NUM, THREAD_NUM * sizeof(int) >> >(gpudata, result);
	
	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);

	int sum[THREAD_NUM * BLOCK_NUM];
	hipMemcpy(&sum, result, sizeof(int)* THREAD_NUM * BLOCK_NUM, hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);
	//-----------------------------------------------
	int final_sum = 0;
	for (int i = 0; i < THREAD_NUM * BLOCK_NUM; i++) {
		final_sum += sum[i];
	}

	float clock_cycle = prop.clockRate * 1e-3f * (float(timeValue) / CLOCKS_PER_SEC);
	float memory_bandwidth = 4 / (float(timeValue) / CLOCKS_PER_SEC); // �u�A�Ω�32�줸��ƫe�� (1024 * 1024 * 32(bit)) / 8(bit -> byte) * 1024(byte -> kb) * 1024(kb -> mb)
	printf("--sum (GPU): %d\n", final_sum);
	printf("--����ɶ� (GPU): %f | �ɯ�: %fMHz | �O�����W�e:%f MB/s\n", float(timeValue) / CLOCKS_PER_SEC, clock_cycle, memory_bandwidth);
}
/*
* ��}�ᥭ��[�`�{��
* shared multiple threads blocks
*/
__global__ static void sumOfSquares_shared_multiple_threads_blocks_continuous_access(int *num, int* result)
{
	extern __shared__ int shared[];
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;

	int i;
	shared[tid] = 0;

	for (i = bid * THREAD_NUM + tid; i < DATA_SIZE;
		i += BLOCK_NUM * THREAD_NUM) {
		shared[tid] += num[i] * num[i];
	}
	__syncthreads();

	if (tid == 0) {
		for (i = 1; i < THREAD_NUM; i++) {
			shared[0] += shared[i];
		}
		result[bid] = shared[0];
	}
}
/*
* Shared Multiple Threads Blocks����
* �s��O����s������
* TreeSum alg.
*/
void ArrayCompute_shared_multiple_threads_blocks_continuous_access_treesum()
{

	float timeValue;
	//-----------------------------------------------
	int* gpudata, *result;
	
	GenerateNumbers(data, DATA_SIZE);
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int)* THREAD_NUM * BLOCK_NUM);
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);

	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);

	sumOfSquares_shared_multiple_threads_blocks_continuous_access_treesum << <BLOCK_NUM, THREAD_NUM, THREAD_NUM * sizeof(int) >> >(gpudata, result);
	
	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);

	int sum[THREAD_NUM * BLOCK_NUM];
	hipMemcpy(&sum, result, sizeof(int)* THREAD_NUM * BLOCK_NUM, hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);
	//-----------------------------------------------
	int final_sum = 0;
	for (int i = 0; i < THREAD_NUM * BLOCK_NUM; i++) {
		final_sum += sum[i];
	}

	float clock_cycle = prop.clockRate * 1e-3f * (float(timeValue) / CLOCKS_PER_SEC);
	float memory_bandwidth = 4 / (float(timeValue) / CLOCKS_PER_SEC); // �u�A�Ω�32�줸��ƫe�� (1024 * 1024 * 32(bit)) / 8(bit -> byte) * 1024(byte -> kb) * 1024(kb -> mb)
	printf("--sum (GPU): %d\n", final_sum);
	printf("--����ɶ� (GPU): %f | �ɯ�: %fMHz | �O�����W�e:%f MB/s\n", float(timeValue) / CLOCKS_PER_SEC, clock_cycle, memory_bandwidth);
}
/*
* ��}�ᥭ��[�`�{��
* shared multiple threads blocks
* TreeSum alg.
*/
__global__ static void sumOfSquares_shared_multiple_threads_blocks_continuous_access_treesum(int *num, int* result)
{
	extern __shared__ int shared[];
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int i;
	int offset = 1, mask = 1;
	shared[tid] = 0;
	for (i = bid * THREAD_NUM + tid; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM) {
		shared[tid] += num[i] * num[i];
	}
	__syncthreads();
	while (offset < THREAD_NUM) {
		if ((tid & mask) == 0) {
			shared[tid] += shared[tid + offset];
		}
		offset += offset;
		mask = offset + mask;
		__syncthreads();
	}
	if (tid == 0) {
		result[bid] = shared[0];
	}
}
/*
* Shared Multiple Threads Blocks����
* �s��O����s������
* ��}TreeSum alg.
*/
void ArrayCompute_shared_multiple_threads_blocks_continuous_access_better_treesum()
{

	float timeValue;
	//-----------------------------------------------
	int* gpudata, *result;
	
	GenerateNumbers(data, DATA_SIZE);
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int)* THREAD_NUM * BLOCK_NUM);
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);

	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);

	sumOfSquares_shared_multiple_threads_blocks_continuous_access_better_treesum << <BLOCK_NUM, THREAD_NUM, THREAD_NUM * sizeof(int) >> >(gpudata, result);
	
	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);

	int sum[THREAD_NUM * BLOCK_NUM];
	hipMemcpy(&sum, result, sizeof(int)* THREAD_NUM * BLOCK_NUM, hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);
	//-----------------------------------------------
	int final_sum = 0;
	for (int i = 0; i < THREAD_NUM * BLOCK_NUM; i++) {
		final_sum += sum[i];
	}

	float clock_cycle = prop.clockRate * 1e-3f * (float(timeValue) / CLOCKS_PER_SEC);
	float memory_bandwidth = 4 / (float(timeValue) / CLOCKS_PER_SEC); // �u�A�Ω�32�줸��ƫe�� (1024 * 1024 * 32(bit)) / 8(bit -> byte) * 1024(byte -> kb) * 1024(kb -> mb)
	printf("--sum (GPU): %d\n", final_sum);
	printf("--����ɶ� (GPU): %f | �ɯ�: %fMHz | �O�����W�e:%f MB/s\n", float(timeValue) / CLOCKS_PER_SEC, clock_cycle, memory_bandwidth);

}
/*
* ��}�ᥭ��[�`�{��
* shared multiple threads blocks
* ��}TreeSum alg.
*/
__global__ static void sumOfSquares_shared_multiple_threads_blocks_continuous_access_better_treesum(int *num, int* result)
{
	extern __shared__ int shared[];
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int i;
	int offset = 1, mask = 1;
	shared[tid] = 0;
	for (i = bid * THREAD_NUM + tid; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM) {
		shared[tid] += num[i] * num[i];
	}
	__syncthreads();

	if (tid < 128) { shared[tid] += shared[tid + 128]; }
	__syncthreads();
	if (tid < 64) { shared[tid] += shared[tid + 64]; }
	__syncthreads();
	if (tid < 32) { shared[tid] += shared[tid + 32]; }
	__syncthreads();
	if (tid < 16) { shared[tid] += shared[tid + 16]; }
	__syncthreads();
	if (tid < 8) { shared[tid] += shared[tid + 8]; }
	__syncthreads();
	if (tid < 4) { shared[tid] += shared[tid + 4]; }
	__syncthreads();
	if (tid < 2) { shared[tid] += shared[tid + 2]; }
	__syncthreads();
	if (tid < 1) { shared[tid] += shared[tid + 1]; }
	__syncthreads();

	if (tid == 0) {
		result[bid] = shared[0];
	}

}


///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////                                        //////////////////////////////////////////
/////////////////////////////////////                  �B�I��                //////////////////////////////////////////
/////////////////////////////////////                 �x�}�ۭ�               //////////////////////////////////////////
/////////////////////////////////////                                        //////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

/*
* �ͦ��üƯB�I�Ưx�}
*/
void matgen(float* a, int lda, int n)
{
	int i, j;
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			a[i * lda + j] = (float)rand() / RAND_MAX + (float)rand() / (RAND_MAX * RAND_MAX);
		}
	}
}

/*
* CPU���x�}���k
*/
void matmult(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n)
{
	int i, j, k;
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			double t = 0;
			for (k = 0; k < n; k++) {
				t += a[i * lda + k] * b[k * ldb + j];
			}
			c[i * ldc + j] = t;
		}
	}
}

/*
* CPU���үx�}�ۭ����G
* �L�X�̤j�~�t�ȡA�����~�t��
*/
void compare_mat(const float* a, int lda, const float* b, int ldb, int n)
{
	float max_err = 0;
	float average_err = 0;
	int i, j;
	for (i = 0; i < n; i++) {
		for (j = 0; j < n; j++) {
			if (b[i * ldb + j] != 0) {
				float err = fabs((a[i * lda + j] - b[i * ldb + j]) / b[i * ldb + j]);
				if (max_err < err) max_err = err; average_err += err;
			}
		}
	}
	printf("�̤j�~�t��: %g �����~�t��: %g\n", max_err, average_err / (n * n));

}

/*
*�B�I�Ưx�}���k
*/
void FloatArrayMultiCompute(int n)
{
	float *a, *b, *c, *d;
	float cpu_time_use_per_sec = 0.0;
	a = (float*)malloc(sizeof(float)* n * n);
	b = (float*)malloc(sizeof(float)* n * n);
	c = (float*)malloc(sizeof(float)* n * n);
	d = (float*)malloc(sizeof(float)* n * n);
	srand(0);
	matgen(a, n, n);
	matgen(b, n, n);
	clock_t time = matmultCUDA(a, n, b, n, c, n, n);
	clock_t cpu_time_start = clock();//CPU�p�ɶ}�l
	matmult(a, n, b, n, d, n, n);
	cpu_time_use_per_sec = (float)(clock() - cpu_time_start) / CLOCKS_PER_SEC;//CPU�p�ɵ����õ���
	compare_mat(c, n, d, n, n);
	double sec = (double)time / CLOCKS_PER_SEC;
	printf("CPU Time used: %f\n", cpu_time_use_per_sec);
	printf("GPU Time used: %f (%f GFLOPS)\n", sec, 2.0 * n * n * n / (sec * 1E9));
}

/*
 * ���I���ө��լ�����o��Ѯv�n�S�a��hipMalloc�BcudaMemcpy2D�BcudaFree�Ҽ{�i�h�p��ɶ��A���e�N���Ҽ{...
 */
clock_t matmultCUDA(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n)
{
	float timeValue;
	float *ac, *bc, *cc;

	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);

	hipMalloc((void**)&ac, sizeof(float)* n * n);
	hipMalloc((void**)&bc, sizeof(float)* n * n);
	hipMalloc((void**)&cc, sizeof(float)* n * n);
	hipMemcpy2D(ac, sizeof(float)* n, a, sizeof(float)* lda, sizeof(float)* n, n, hipMemcpyHostToDevice);
	hipMemcpy2D(bc, sizeof(float)* n, b, sizeof(float)* ldb, sizeof(float)* n, n, hipMemcpyHostToDevice);

	int blocks = (n + THREAD_NUM - 1) / THREAD_NUM;
	matMultCUDA << <blocks * n, THREAD_NUM >> >(ac, n, bc, n, cc, n, n);
	hipMemcpy2D(c, sizeof(float)* ldc, cc, sizeof(float)* n, sizeof(float)* n, n, hipMemcpyDeviceToHost);
	hipFree(ac);
	hipFree(bc);
	hipFree(cc);

	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);


	return timeValue;
}

/*
 * �x�}�ۭ���Kernel�禡
 */
__global__ static void matMultCUDA(const float* a, size_t lda,const float* b, size_t ldb, float* c, size_t ldc, int n)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	const int idx = bid * blockDim.x + tid;
	const int row = idx / n;
	const int column = idx % n;
	int i;
	if (row < n && column < n) {
		float t = 0;
		for (i = 0; i < n; i++) {
			t += a[row * lda + i] * b[i * ldb + column];
		}
		c[row * ldc + column] = t;
	}
}

/*
 * �B�I�Ưx�}���k
 * Kahan'sSummation Formula
 */
void FloatArrayMultiCompute_KSF(int n)
{
	float *a, *b, *c, *d;
	a = (float*)malloc(sizeof(float)* n * n);
	b = (float*)malloc(sizeof(float)* n * n);
	c = (float*)malloc(sizeof(float)* n * n);
	d = (float*)malloc(sizeof(float)* n * n);
	srand(0);
	matgen(a, n, n);
	matgen(b, n, n);
	clock_t time = matmultCUDA_KSF(a, n, b, n, c, n, n);
	matmult(a, n, b, n, d, n, n);
	compare_mat(c, n, d, n, n);
	double sec = (double)time / CLOCKS_PER_SEC;
	printf("Time used: %f (%f GFLOPS)\n", sec, 2.0 * n * n * n / (sec * 1E9));
}

/*
* ���I���ө��լ�����o��Ѯv�n�S�a��hipMalloc�BcudaMemcpy2D�BcudaFree�Ҽ{�i�h�p��ɶ��A���e�N���Ҽ{...
* Kahan'sSummation Formula
*/
clock_t matmultCUDA_KSF(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n)
{
	float timeValue;
	float *ac, *bc, *cc;

	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);

	hipMalloc((void**)&ac, sizeof(float)* n * n);
	hipMalloc((void**)&bc, sizeof(float)* n * n);
	hipMalloc((void**)&cc, sizeof(float)* n * n);
	hipMemcpy2D(ac, sizeof(float)* n, a, sizeof(float)* lda, sizeof(float)* n, n, hipMemcpyHostToDevice);
	hipMemcpy2D(bc, sizeof(float)* n, b, sizeof(float)* ldb, sizeof(float)* n, n, hipMemcpyHostToDevice);

	int blocks = (n + THREAD_NUM - 1) / THREAD_NUM;
	matMultCUDA_KSF << <blocks * n, THREAD_NUM >> >(ac, n, bc, n, cc, n, n);
	hipMemcpy2D(c, sizeof(float)* ldc, cc, sizeof(float)* n, sizeof(float)* n, n, hipMemcpyDeviceToHost);
	hipFree(ac);
	hipFree(bc);
	hipFree(cc);

	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);


	return timeValue;
}

/*
* �x�}�ۭ���Kernel�禡
* Kahan'sSummation Formula
*/
__global__ static void matMultCUDA_KSF(const float* a, size_t lda, const float* b, size_t ldb, float* c, size_t ldc, int n)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	const int idx = bid * blockDim.x + tid;
	const int row = idx / n;
	const int column = idx % n;
	int i;
	if (row < n && column < n) {
		float t = 0;
		float y = 0;
		for (i = 0; i < n; i++) {
			//�o��u���ݤ����b�F��...
			float r;
			y -= a[row * lda + i] * b[i * ldb + column];
			r = t - y;
			y = (r - t) + y;
			t = r;
		}
		c[row * ldc + column] = t; //�o��O�ڥ[�W�h���A�]�����[�o�����~�t�ȳ��|�O1�A�D�n���D�I�٬O�W��...
	}
	//�o�q�g�k����n�z�ѡA�i�O��Ѯv�o�y�L���P...
	/*if (row < n && column < n) {
		float sum = 0;
		float z = 0;
		for (i = 0; i < n; i++) {
			float y = a[row * lda + i] * b[i * ldb + column] - z;
			float t = sum + y;
			z = (t - sum) - y;
			sum += t;
		}
		c[row * ldc + column] = sum;
	}*/
}


/*
* �B�I�Ưx�}���k
* Kahan'sSummation Formula
* Shared memory
* Pitch
*/
void FloatArrayMultiCompute_KSF_shared_pitch(int n)
{
	float *a, *b, *c, *d;
	a = (float*)malloc(sizeof(float)* n * n);
	b = (float*)malloc(sizeof(float)* n * n);
	c = (float*)malloc(sizeof(float)* n * n);
	d = (float*)malloc(sizeof(float)* n * n);
	srand(0);
	matgen(a, n, n);
	matgen(b, n, n);
	clock_t time = matmultCUDA_KSF_shared_pitch(a, n, b, n, c, n, n);
	matmult(a, n, b, n, d, n, n);
	compare_mat(c, n, d, n, n);
	double sec = (double)time / CLOCKS_PER_SEC;
	printf("Time used: %f (%f GFLOPS)\n", sec, 2.0 * n * n * n / (sec * 1E9));
}

/*
* ���I���ө��լ�����o��Ѯv�n�S�a��hipMalloc�BcudaMemcpy2D�BcudaFree�Ҽ{�i�h�p��ɶ��A���e�N���Ҽ{...
* Kahan'sSummation Formula
* Shared memory
* Pitch
*/
clock_t matmultCUDA_KSF_shared_pitch(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n)
{
	float timeValue;
	float *ac, *bc, *cc;

	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);

	size_t pitch_a, pitch_b, pitch_c;

	hipMallocPitch((void**)&ac, &pitch_a, sizeof(float)* n, n);
	hipMallocPitch((void**)&bc, &pitch_b, sizeof(float)* n, n);
	hipMallocPitch((void**)&cc, &pitch_c, sizeof(float)* n, n);
	hipMemcpy2D(ac, pitch_a, a, sizeof(float)* lda,sizeof(float)* n, n, hipMemcpyHostToDevice);
	hipMemcpy2D(bc, pitch_b, b, sizeof(float)* ldb,sizeof(float)* n, n, hipMemcpyHostToDevice);

	int blocks = (n + THREAD_NUM - 1) / THREAD_NUM;
	matMultCUDA_KSF_shared_pitch << <n, THREAD_NUM, sizeof(float)* n >> >(ac, pitch_a / sizeof(float), bc, pitch_b / sizeof(float),cc, pitch_c / sizeof(float), n);
	hipMemcpy2D(c, sizeof(float)* ldc, cc, pitch_c,sizeof(float)* n, n, hipMemcpyDeviceToHost);
	hipFree(ac);
	hipFree(bc);
	hipFree(cc);

	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);


	return timeValue;
}

/*
* �x�}�ۭ���Kernel�禡
* Kahan'sSummation Formula
* Shared memory
* Pitch
*/
__global__ static void matMultCUDA_KSF_shared_pitch(const float* a, size_t lda,const float* b, size_t ldb, float* c, size_t ldc, int n)
{
	extern __shared__ float shared_data[];
	const int tid = threadIdx.x;
	const int row = blockIdx.x;
	int i, j;

	for (i = tid; i < n; i += blockDim.x) {
		shared_data[i] = a[row * lda + i];
	}

	__syncthreads();

	for (j = tid; j < n; j += blockDim.x) {
		float t = 0;
		float y = 0;
		for (i = 0; i < n; i++) {
			float r;
			y -= shared_data[i] * b[i * ldb + j];
			r = t - y;
			y = (r - t) + y;
			t = r;
		}
		c[row * ldc + j] = t;
	}
}

/*
 * �ϯx�}
 */
void inverse_matrix(int n)
{
	int i, j;
	int* a = new int[n*n];
	float determinant = 0;
	printf("Find Inverse Of Matrix by Subham Mishra\n");
	printf("Enter elements of n x n matrix:\n");
	for (i = 0; i<n; i++)
	{
		for (j = 0; j<n; j++)
		{
			int num;
			scanf("%d", &num);
			a[i*j+j] = num;
		}
	}
	printf("The entered matrix is:\n");
	for (i = 0; i<n; i++)
	{
		for (j = 0; j<n; j++)
		{
			printf("%d\n", a[i*j + j]);
		}
		
	}
	for (i = 0; i<n; i++)
	{
		determinant = determinant + (a[0][i] * (a[1][(i + 1) % n] *
			a[2][(i + 2) % n] - a[1][(i + 2) % n] * a[2][(i + 1) % n]));
	}
	if (determinant == 0)
	{
		printf("Inverse does not exist (Determinant=0).\n");
	}
	else
	{
		printf("Inverse of matrix is: \n");
	}
	for (i = 0; i<n; i++)
	{
		for (j = 0; j<n; j++)
		{
			printf("%f\t", (float)(a[(i + 1) % n][(j + 1) % n] *
				a[(i + 2) % n][(j + 2) % n]) - (a[(i + 1) % n][(j + 2) % n] *
				a[(i + 2) % n][(j + 1) % n]) / determinant);
		}
		printf("\n");
	}
}
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <ostream>
#include <iostream>
#define DATA_SIZE 1048576
#define THREAD_NUM 256

int data[DATA_SIZE];
bool InitCUDA();
void GenerateNumbers(int *number, int size);
void ArrayCompute();
void ArrayCompute_multiple_threads();
__global__ static void sumOfSquares(int *num, int* result);
__global__ static void sumOfSquares_multiple_threads(int *num, int* result);
hipDeviceProp_t prop;

int main()
{
	if (InitCUDA())
	{
		ArrayCompute();
		printf("==================256Threads��}��==================\n");
		ArrayCompute_multiple_threads();
	}
	

	printf("\nDone!");
	getchar();
	return 0;
}

bool InitCUDA()
{
	int count;

	hipGetDeviceCount(&count);
	if (count == 0)
	{
		fprintf(stderr, "No Device!!");
		return false;
	}

	int i;
	for (i = 0; i < count; i++)
	{
		
		if (hipGetDeviceProperties(&prop, i) == hipSuccess)
		{
			if (prop.major >= 1)
			{
				printf("��%d�i��d�W�� -> %s\n", count, prop.name);
				printf("--CUDA���� -> %d.%d\n", prop.major, prop.minor);
				char msg[256];
				sprintf_s(msg, "--�`Global Memory -> %.0f MBytes (%llu bytes)\n",
					(float)prop.totalGlobalMem / 1048576.0f, (unsigned long long) prop.totalGlobalMem);
				printf("%s", msg);
				printf("--%2d �� Multiprocessors\n", prop.multiProcessorCount);
				printf("--�C��Multiprocessors�̳̤j������ƶq:%d\n", prop.maxThreadsPerMultiProcessor);
				printf("--�C��Block�̳̤j������ƶq:%d\n", prop.maxThreadsPerBlock);
				printf("--GPU �̤j�ɯ�: %.0f MHz (%0.2f GHz)\n", prop.clockRate * 1e-3f, prop.clockRate * 1e-6f);

				break;
			}
		}
	}

	if (i == count)
	{
		fprintf(stderr, "No device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}

void GenerateNumbers(int *number, int size)
{
	for (int i = 0; i < size; i++)
	{
		number[i] = rand() % 10;
	}
}

void ArrayCompute()
{
	float timeValue;


	//-----------------------------------------------
	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);
	GenerateNumbers(data, DATA_SIZE);
	int* gpudata, *result;
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int));
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);
	sumOfSquares << <1, 1, 0 >> >(gpudata, result);
	int sum;
	hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);
	printf("--sum (GPU): %d\n", sum);
	printf("--����ɶ� (GPU): %f\n", float(timeValue) / CLOCKS_PER_SEC);
	//-----------------------------------------------

	sum = 0;
	clock_t cpu_time = clock();
	for (int i = 0; i < DATA_SIZE; i++) {
		sum += data[i] * data[i];
	}
	printf("--sum (CPU): %d\n", sum);
	printf("--����ɶ� (CPU): %f\n", float(clock() - cpu_time) / CLOCKS_PER_SEC);
}

void ArrayCompute_multiple_threads()
{
	float timeValue;
	//-----------------------------------------------
	int* gpudata, *result;
	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);
	GenerateNumbers(data, DATA_SIZE);
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int)* THREAD_NUM);
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);

	sumOfSquares_multiple_threads << <1, THREAD_NUM, 0 >> >(gpudata, result);

	int sum[THREAD_NUM];
	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);
	hipMemcpy(&sum, result, sizeof(int)* THREAD_NUM, hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);

	//-----------------------------------------------
	int final_sum = 0;
	for (int i = 0; i < THREAD_NUM; i++) {
		final_sum += sum[i];
	}
	float clock_cycle = prop.clockRate * 1e-3f * (float(timeValue) / CLOCKS_PER_SEC); 
	float memory_bandwidth = 4 / (float(timeValue) / CLOCKS_PER_SEC); // �u�A�Ω�32�줸��ƫe�� (1024 * 1024 * 32(bit)) / 8(bit -> byte) * 1024(byte -> kb) * 1024(kb -> mb)
	printf("sum (GPU): %d\n", final_sum);
	printf("����ɶ� (GPU): %f �ɯ�: %fMHz �O�����W�e:%f MB/s\n", float(timeValue) / CLOCKS_PER_SEC, clock_cycle, memory_bandwidth);

	final_sum = 0;
	clock_t cpu_time = clock();
	for (int i = 0; i < DATA_SIZE; i++) {
		final_sum += data[i] * data[i];
	}
	printf("sum (CPU): %d\n", final_sum);
	printf("����ɶ� (CPU): %f\n", float(clock() - cpu_time) / CLOCKS_PER_SEC);
}

/*
* �쪩����[�`�{��
*/
__global__ static void sumOfSquares(int *num, int* result)
{
	int sum = 0;
	int i;
	for (i = 0; i < DATA_SIZE; i++) {
		sum += num[i] * num[i];
	}
	*result = sum;
}

__global__ static void sumOfSquares_multiple_threads(int *num, int* result)
{
	const int tid = threadIdx.x;
	const int size = DATA_SIZE / THREAD_NUM;
	int sum = 0;
	int i;
	for (i = tid * size; i < (tid + 1) * size; i++) {
		sum += num[i] * num[i];
	}
	result[tid] = sum;
}
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <ostream>
#include <iostream>
#define DATA_SIZE 1048576
#define BLOCK_NUM 32
#define THREAD_NUM 256

int data[DATA_SIZE];
bool InitCUDA();
void GenerateNumbers(int *number, int size);
void ArrayCompute();
void ArrayCompute_multiple_threads();
void ArrayCompute_multiple_threads_continuous_access();
void ArrayCompute_multiple_threads_blocks_continuous_access();
void ArrayCompute_shared_multiple_threads_blocks_continuous_access();
void ArrayCompute_shared_multiple_threads_blocks_continuous_access_treesum();
void ArrayCompute_shared_multiple_threads_blocks_continuous_access_better_treesum();
__global__ static void sumOfSquares(int *num, int* result);
__global__ static void sumOfSquares_multiple_threads(int *num, int* result);
__global__ static void sumOfSquares_multiple_threads_continuous_access(int *num, int* result);
__global__ static void sumOfSquares_multiple_threads_blocks_continuous_access(int *num, int* result);
__global__ static void sumOfSquares_shared_multiple_threads_blocks_continuous_access(int *num, int* result);
__global__ static void sumOfSquares_shared_multiple_threads_blocks_continuous_access_treesum(int *num, int* result);
__global__ static void sumOfSquares_shared_multiple_threads_blocks_continuous_access_better_treesum(int *num, int* result);
hipDeviceProp_t prop;

int main()
{
	if (InitCUDA())
	{
		ArrayCompute();
		printf("(1).%dThreads��}����\n", THREAD_NUM);
		ArrayCompute_multiple_threads();
		printf("(2).(3).%dThreads �s��O����s������\n", THREAD_NUM);
		ArrayCompute_multiple_threads_continuous_access();
		printf("(4).%dThreads %dBlocks �s��O����s������\n", THREAD_NUM, BLOCK_NUM);
		ArrayCompute_multiple_threads_blocks_continuous_access();
		printf("(5).Shared Memory %dThreads %dBlocks �s��O����s������\n", THREAD_NUM, BLOCK_NUM);
		ArrayCompute_shared_multiple_threads_blocks_continuous_access();
		printf("(6).TreeSum alg. Shared Memory %dThreads %dBlocks �s��O����s������\n", THREAD_NUM, BLOCK_NUM);
		ArrayCompute_shared_multiple_threads_blocks_continuous_access_treesum();
		printf("(7).��}TreeSum alg. Shared Memory %dThreads %dBlocks �s��O����s������\n", THREAD_NUM, BLOCK_NUM);
		ArrayCompute_shared_multiple_threads_blocks_continuous_access_better_treesum();
	}
	

	printf("\nDone!");
	getchar();
	return 0;
}

bool InitCUDA()
{
	int count;

	hipGetDeviceCount(&count);
	if (count == 0)
	{
		fprintf(stderr, "No Device!!");
		return false;
	}

	int i;
	for (i = 0; i < count; i++)
	{
		
		if (hipGetDeviceProperties(&prop, i) == hipSuccess)
		{
			if (prop.major >= 1)
			{
				printf("��%d�i��d�W�� -> %s\n", count, prop.name);
				printf("--CUDA���� -> %d.%d\n", prop.major, prop.minor);
				char msg[256];
				sprintf_s(msg, "--�`Global Memory -> %.0f MBytes (%llu bytes)\n",
					(float)prop.totalGlobalMem / 1048576.0f, (unsigned long long) prop.totalGlobalMem);
				printf("%s", msg);
				printf("--%2d �� Multiprocessors\n", prop.multiProcessorCount);
				printf("--�C��Multiprocessors�̳̤j������ƶq:%d\n", prop.maxThreadsPerMultiProcessor);
				printf("--�C��Block�̳̤j������ƶq:%d\n", prop.maxThreadsPerBlock);
				printf("--GPU �̤j�ɯ�: %.0f MHz (%0.2f GHz)\n", prop.clockRate * 1e-3f, prop.clockRate * 1e-6f);

				break;
			}
		}
	}

	if (i == count)
	{
		fprintf(stderr, "No device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}

void GenerateNumbers(int *number, int size)
{
	for (int i = 0; i < size; i++)
	{
		number[i] = rand() % 10;
	}
}
/*
* 1 Thread����
*/
void ArrayCompute()
{
	float timeValue;


	//-----------------------------------------------
	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);
	GenerateNumbers(data, DATA_SIZE);
	int* gpudata, *result;
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int));
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);
	sumOfSquares << <1, 1, 0 >> >(gpudata, result);
	int sum;
	hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);
	printf("--sum (GPU): %d\n", sum);
	printf("--����ɶ� (GPU): %f\n", float(timeValue) / CLOCKS_PER_SEC);
	//-----------------------------------------------

	sum = 0;
	clock_t cpu_time = clock();
	for (int i = 0; i < DATA_SIZE; i++) {
		sum += data[i] * data[i];
	}
	printf("--sum (CPU): %d\n", sum);
	printf("--����ɶ� (CPU): %f\n", float(clock() - cpu_time) / CLOCKS_PER_SEC);

}

/*
 * Multiple Threads����
 */
void ArrayCompute_multiple_threads()
{
	float timeValue;
	//-----------------------------------------------
	int* gpudata, *result;
	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);
	GenerateNumbers(data, DATA_SIZE);
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int)* THREAD_NUM);
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);

	sumOfSquares_multiple_threads << <1, THREAD_NUM, 0 >> >(gpudata, result);

	int sum[THREAD_NUM];
	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);
	hipMemcpy(&sum, result, sizeof(int)* THREAD_NUM, hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);

	//-----------------------------------------------
	int final_sum = 0;
	for (int i = 0; i < THREAD_NUM; i++) {
		final_sum += sum[i];
	}
	float clock_cycle = prop.clockRate * 1e-3f * (float(timeValue) / CLOCKS_PER_SEC); 
	float memory_bandwidth = 4 / (float(timeValue) / CLOCKS_PER_SEC); // �u�A�Ω�32�줸��ƫe�� (1024 * 1024 * 32(bit)) / 8(bit -> byte) * 1024(byte -> kb) * 1024(kb -> mb)
	printf("--sum (GPU): %d\n", final_sum);
	printf("--����ɶ� (GPU): %f | �ɯ�: %fMHz | �O�����W�e:%f MB/s\n", float(timeValue) / CLOCKS_PER_SEC, clock_cycle, memory_bandwidth);
}

/*
* Multiple Threads����
* �s��O����s������
*/
void ArrayCompute_multiple_threads_continuous_access()
{
	float timeValue;
	//-----------------------------------------------
	int* gpudata, *result;
	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);
	GenerateNumbers(data, DATA_SIZE);
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int)* THREAD_NUM);
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);

	sumOfSquares_multiple_threads_continuous_access << <1, THREAD_NUM, 0 >> >(gpudata, result);

	int sum[THREAD_NUM];
	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);
	hipMemcpy(&sum, result, sizeof(int)* THREAD_NUM, hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);

	//-----------------------------------------------
	int final_sum = 0;
	for (int i = 0; i < THREAD_NUM; i++) {
		final_sum += sum[i];
	}
	float clock_cycle = prop.clockRate * 1e-3f * (float(timeValue) / CLOCKS_PER_SEC);
	float memory_bandwidth = 4 / (float(timeValue) / CLOCKS_PER_SEC); // �u�A�Ω�32�줸��ƫe�� (1024 * 1024 * 32(bit)) / 8(bit -> byte) * 1024(byte -> kb) * 1024(kb -> mb)
	printf("--sum (GPU): %d\n", final_sum);
	printf("--����ɶ� (GPU): %f | �ɯ�: %fMHz | �O�����W�e:%f MB/s\n", float(timeValue) / CLOCKS_PER_SEC, clock_cycle, memory_bandwidth);
}

/*
* Multiple Threads Blocks����
* �s��O����s������
*/
void ArrayCompute_multiple_threads_blocks_continuous_access()
{
	
	float timeValue;
	//-----------------------------------------------
	int* gpudata, *result;
	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);
	GenerateNumbers(data, DATA_SIZE);
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int)* THREAD_NUM * BLOCK_NUM);
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);
	sumOfSquares_multiple_threads_blocks_continuous_access << <BLOCK_NUM, THREAD_NUM, 0 >> >(gpudata, result);
	int sum[THREAD_NUM * BLOCK_NUM];
	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);
	hipMemcpy(&sum, result, sizeof(int)* THREAD_NUM * BLOCK_NUM, hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);
	//-----------------------------------------------
	int final_sum = 0;
	for (int i = 0; i < THREAD_NUM * BLOCK_NUM; i++) {
		final_sum += sum[i];
	}
	
	float clock_cycle = prop.clockRate * 1e-3f * (float(timeValue) / CLOCKS_PER_SEC);
	float memory_bandwidth = 4 / (float(timeValue) / CLOCKS_PER_SEC); // �u�A�Ω�32�줸��ƫe�� (1024 * 1024 * 32(bit)) / 8(bit -> byte) * 1024(byte -> kb) * 1024(kb -> mb)
	printf("--sum (GPU): %d\n", final_sum);
	printf("--����ɶ� (GPU): %f | �ɯ�: %fMHz | �O�����W�e:%f MB/s\n", float(timeValue) / CLOCKS_PER_SEC, clock_cycle, memory_bandwidth);
}

/*
* Shared Multiple Threads Blocks����
* �s��O����s������
*/
void ArrayCompute_shared_multiple_threads_blocks_continuous_access()
{

	float timeValue;
	//-----------------------------------------------
	int* gpudata, *result;
	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);
	GenerateNumbers(data, DATA_SIZE);
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int)* THREAD_NUM * BLOCK_NUM);
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);
	sumOfSquares_shared_multiple_threads_blocks_continuous_access << <BLOCK_NUM, THREAD_NUM, THREAD_NUM * sizeof(int) >> >(gpudata, result);
	int sum[THREAD_NUM * BLOCK_NUM];
	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);
	hipMemcpy(&sum, result, sizeof(int)* THREAD_NUM * BLOCK_NUM, hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);
	//-----------------------------------------------
	int final_sum = 0;
	for (int i = 0; i < THREAD_NUM * BLOCK_NUM; i++) {
		final_sum += sum[i];
	}

	float clock_cycle = prop.clockRate * 1e-3f * (float(timeValue) / CLOCKS_PER_SEC);
	float memory_bandwidth = 4 / (float(timeValue) / CLOCKS_PER_SEC); // �u�A�Ω�32�줸��ƫe�� (1024 * 1024 * 32(bit)) / 8(bit -> byte) * 1024(byte -> kb) * 1024(kb -> mb)
	printf("--sum (GPU): %d\n", final_sum);
	printf("--����ɶ� (GPU): %f | �ɯ�: %fMHz | �O�����W�e:%f MB/s\n", float(timeValue) / CLOCKS_PER_SEC, clock_cycle, memory_bandwidth);
}

/*
* Shared Multiple Threads Blocks����
* �s��O����s������
* TreeSum alg.
*/
void ArrayCompute_shared_multiple_threads_blocks_continuous_access_treesum()
{

	float timeValue;
	//-----------------------------------------------
	int* gpudata, *result;
	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);
	GenerateNumbers(data, DATA_SIZE);
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int)* THREAD_NUM * BLOCK_NUM);
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);
	sumOfSquares_shared_multiple_threads_blocks_continuous_access_treesum << <BLOCK_NUM, THREAD_NUM, THREAD_NUM * sizeof(int) >> >(gpudata, result);
	int sum[THREAD_NUM * BLOCK_NUM];
	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);
	hipMemcpy(&sum, result, sizeof(int)* THREAD_NUM * BLOCK_NUM, hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);
	//-----------------------------------------------
	int final_sum = 0;
	for (int i = 0; i < THREAD_NUM * BLOCK_NUM; i++) {
		final_sum += sum[i];
	}

	float clock_cycle = prop.clockRate * 1e-3f * (float(timeValue) / CLOCKS_PER_SEC);
	float memory_bandwidth = 4 / (float(timeValue) / CLOCKS_PER_SEC); // �u�A�Ω�32�줸��ƫe�� (1024 * 1024 * 32(bit)) / 8(bit -> byte) * 1024(byte -> kb) * 1024(kb -> mb)
	printf("--sum (GPU): %d\n", final_sum);
	printf("--����ɶ� (GPU): %f | �ɯ�: %fMHz | �O�����W�e:%f MB/s\n", float(timeValue) / CLOCKS_PER_SEC, clock_cycle, memory_bandwidth);
}

/*
* Shared Multiple Threads Blocks����
* �s��O����s������
* ��}TreeSum alg.
*/
void ArrayCompute_shared_multiple_threads_blocks_continuous_access_better_treesum()
{

	float timeValue;
	//-----------------------------------------------
	int* gpudata, *result;
	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate(&beginEvent);
	hipEventCreate(&endEvent);
	hipEventRecord(beginEvent, 0);
	GenerateNumbers(data, DATA_SIZE);
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int)* THREAD_NUM * BLOCK_NUM);
	hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);
	sumOfSquares_shared_multiple_threads_blocks_continuous_access_better_treesum << <BLOCK_NUM, THREAD_NUM, THREAD_NUM * sizeof(int) >> >(gpudata, result);
	int sum[THREAD_NUM * BLOCK_NUM];
	hipEventRecord(endEvent, 0);
	hipEventSynchronize(endEvent);
	hipEventElapsedTime(&timeValue, beginEvent, endEvent);
	hipEventDestroy(beginEvent);
	hipEventDestroy(endEvent);
	hipMemcpy(&sum, result, sizeof(int)* THREAD_NUM * BLOCK_NUM, hipMemcpyDeviceToHost);
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);
	//-----------------------------------------------
	int final_sum = 0;
	for (int i = 0; i < THREAD_NUM * BLOCK_NUM; i++) {
		final_sum += sum[i];
	}

	float clock_cycle = prop.clockRate * 1e-3f * (float(timeValue) / CLOCKS_PER_SEC);
	float memory_bandwidth = 4 / (float(timeValue) / CLOCKS_PER_SEC); // �u�A�Ω�32�줸��ƫe�� (1024 * 1024 * 32(bit)) / 8(bit -> byte) * 1024(byte -> kb) * 1024(kb -> mb)
	printf("--sum (GPU): %d\n", final_sum);
	printf("--����ɶ� (GPU): %f | �ɯ�: %fMHz | �O�����W�e:%f MB/s\n", float(timeValue) / CLOCKS_PER_SEC, clock_cycle, memory_bandwidth);

}

/*
* �쪩����[�`�{��
*/
__global__ static void sumOfSquares(int *num, int* result)
{
	int sum = 0;
	int i;
	for (i = 0; i < DATA_SIZE; i++) {
		sum += num[i] * num[i];
	}
	*result = sum;
}

/*
* ��}�ᥭ��[�`�{��
* multiple threads blocks
*/
__global__ static void sumOfSquares_multiple_threads(int *num, int* result)
{
	const int tid = threadIdx.x;
	const int size = DATA_SIZE / THREAD_NUM;
	int sum = 0;
	int i;
	for (i = tid * size; i < (tid + 1) * size; i++) {
		sum += num[i] * num[i];
	}
	result[tid] = sum;
}
/*
* ��}�ᥭ��[�`�{��
* multiple threads �s��O����s��
*/
__global__ static void sumOfSquares_multiple_threads_continuous_access(int *num, int* result)
{
	const int tid = threadIdx.x;
	int sum = 0;
	int i;
	for (i = tid; i < DATA_SIZE; i += THREAD_NUM) {
		sum += num[i] * num[i];
	}
	result[tid] = sum;
}
/*
* ��}�ᥭ��[�`�{��
* multiple threads blocks
*/
__global__ static void sumOfSquares_multiple_threads_blocks_continuous_access(int *num, int* result)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int sum = 0;
	int i;
	for (i = bid * THREAD_NUM + tid; i < DATA_SIZE;i += BLOCK_NUM * THREAD_NUM) {
		sum += num[i] * num[i];
	}
	result[bid * THREAD_NUM + tid] = sum;
}

/*
* ��}�ᥭ��[�`�{��
* shared multiple threads blocks
*/
__global__ static void sumOfSquares_shared_multiple_threads_blocks_continuous_access(int *num, int* result)
{
	extern __shared__ int shared[];
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;

	int i;
	shared[tid] = 0;

	for (i = bid * THREAD_NUM + tid; i < DATA_SIZE;
		i += BLOCK_NUM * THREAD_NUM) {
		shared[tid] += num[i] * num[i];
	}
	__syncthreads();

	if (tid == 0) {
		for (i = 1; i < THREAD_NUM; i++) {
			shared[0] += shared[i];
		}
		result[bid] = shared[0];
	}
}

/*
* ��}�ᥭ��[�`�{��
* shared multiple threads blocks
* TreeSum alg.
*/
__global__ static void sumOfSquares_shared_multiple_threads_blocks_continuous_access_treesum(int *num, int* result)
{
	extern __shared__ int shared[];
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int i;
	int offset = 1, mask = 1;
	shared[tid] = 0;
	for (i = bid * THREAD_NUM + tid; i < DATA_SIZE;i += BLOCK_NUM * THREAD_NUM) {
		shared[tid] += num[i] * num[i];
	}
	__syncthreads();
	while (offset < THREAD_NUM) {
		if ((tid & mask) == 0) {
			shared[tid] += shared[tid + offset];
		}
		offset += offset;
		mask = offset + mask;
		__syncthreads();
	}
	if (tid == 0) {
		result[bid] = shared[0];
	}
}

/*
* ��}�ᥭ��[�`�{��
* shared multiple threads blocks
* ��}TreeSum alg.
*/
__global__ static void sumOfSquares_shared_multiple_threads_blocks_continuous_access_better_treesum(int *num, int* result)
{
	extern __shared__ int shared[];
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int i;
	int offset = 1, mask = 1;
	shared[tid] = 0;
	for (i = bid * THREAD_NUM + tid; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM) {
		shared[tid] += num[i] * num[i];
	}
	__syncthreads();
	
	if (tid < 128) { shared[tid] += shared[tid + 128]; }
	__syncthreads();
	if (tid < 64) { shared[tid] += shared[tid + 64]; }
	__syncthreads();
	if (tid < 32) { shared[tid] += shared[tid + 32]; }
	__syncthreads();
	if (tid < 16) { shared[tid] += shared[tid + 16]; }
	__syncthreads();
	if (tid < 8) { shared[tid] += shared[tid + 8]; }
	__syncthreads();
	if (tid < 4) { shared[tid] += shared[tid + 4]; }
	__syncthreads();
	if (tid < 2) { shared[tid] += shared[tid + 2]; }
	__syncthreads();
	if (tid < 1) { shared[tid] += shared[tid + 1]; }
	__syncthreads();
	
	if (tid == 0) {
		result[bid] = shared[0];
	}
}
